#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common/book.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <locale.h>

#ifdef LINUX
#include <getopt.h>
#include <sys/time.h>
#include <unistd.h>
#include <wait.h>
#else
#include <ctype.h>
#define __GNU_LIBRARY__

#define MESTRADOCUDA
#include "gettimeofday_win.h"
#include "getopt.h"

#endif

#define PI 3.14159265
#define G 9.806

float hr; 
float f;
float l;            // Tamanho horizontal do pipe em metros
float d;            // Diametro do pipe
float dt;           // Delta T 
float tmax;         // Tempo máximo da Simulação em seungos (Parametro bom pra mexer)

float cdao;

float r;
float a;            // Celeridade da Onda
float qo;
float b;
float cm;
float cp;
float dx;           // Delta X (Comprimento do tubo pelo número de seguimentos)
float tf;

float as;
float t;
float bp;
float bm;

struct _HQT {
    float H;
    float Q;
    float T;
};

struct _HQ {
    float H;
    float Q;
};

struct _HQT *matrizHQT;           //Vetor da Strutura H (Pressão) Q(Vazão) e T(Tempo)
struct _HQT *_dMatrizHQT;         //Matriz da Strutura H (Pressão) Q(Vazão) e T(Tempo) em device

struct _HQ *atualHQ;
struct _HQ *proximolHQ;

unsigned int n;              //Número de segmentos
unsigned int N;              //Numeros de segmentos * Delta T
unsigned int ns;             // Numero de segmentos + segmentos montante + segmentos justante
unsigned int linha;

struct ConstantesPrograma {
    unsigned int ns;
    float qo;
    float hr;
    float r;
    float b;
    float t;
    float dt;
    float tmax;
    unsigned int controle;
    unsigned int nthreads;

    __device__ unsigned int get_ns() { return ns; }
    __device__ float get_qo() { return qo; }
    __device__ float get_hr() { return hr; }
    __device__ float get_r() { return r; }
    __device__ float get_b() { return b; }
    __device__ float get_t() { return t; }
    __device__ float get_dt() { return dt; }
    __device__ float get_tmax() { return tmax; }
    __device__ unsigned int get_controle() { return controle; }
    __device__ unsigned int get_nthreads() { return nthreads; }
};

__constant__ __device__ ConstantesPrograma ConstantDeviceConstantes;

/* Globals set by command line args */
int verbosity = 0; /* print trace if set */
int exibe_tempos = 0;

unsigned int nblocks; //Número de blocos que vão ser chamados
unsigned int nthreads; //Número de threads por blocls que vão ser chamados

unsigned int multiplicador = 1;

int gravarDisco = 0;
char* narq_e = NULL;
char* narq_s = NULL;

char narqe_default[] = "entrada.dat";
char narqs_default[] = "saida.dat";

// NUm Threads
int chunk = 512;
 
enum _modo {SINGLE = 0, MULTI = 1, CUDA = 2};
enum _modo modo = SINGLE;

extern int paralelo = 0;

//Inicializa vetor com ZEROS
void initZero(float *vector, int size) {
    int i;
    for (i = 0; i < size; i++) {
        vector[i] = 0.0;
    }  
}

void initZeroHQT(struct _HQT *vector, int size) {
    int i;
    #pragma omp parallel for if(paralelo)
    for (i = 0; i < size; i++) {
        vector[i].H = 0.0;
        vector[i].Q = 0.0;
        vector[i].T = 0.0;
    }  
}

void initZeroHQ(struct _HQ *vector, int size) {
    int i;
    #pragma omp parallel for if(paralelo)
    for (i = 0; i < size; i++) {
        vector[i].H = 0.0;
        vector[i].Q = 0.0;
    }  
}

/*
 * printUsage - Print usage info
 */
void printUsage(char* argv[]) {
    printf("Uso: %s [-hv] -e <file> -s <file> -m 0|1|2 -t <num> -b <num> -d <num> -p -t\n", argv[0]);
    printf("Options:\n");
    printf("  -h          Print this help message.\n");
    printf("  -v          Optional verbose flag.\n");
    printf("  -e <file>   Arquivo de entrada.\n");
    printf("  -s <file>   Arquivo de saida com o resultado.\n");
    printf("  -m <modo>   0 = Singlecore | 1 = Multicore | 2 = CUDA: Utiliza a versão indicada.\n");
    printf("  -t <num>    Número de Threads.\n");
    printf("  -d <num>    Número de vezes que vai rodar o simulação (Simula o tamanho do problema).\n");
    printf("  -p          Roda as instâncias do problema de forma paralela.\n");
    printf("  -t          Exibe os tempos apenasa.\n");
    
    printf("\nExamples:\n");
    printf("  >  %s -v -e entrada.dat -s resuldado.dat -m 2 -b 16 -t 512 -d 5000\n", argv[0]);
    printf("  >  %s -v -e entrada.dat -s resuldado.dat -m 1 -c 512 -d 5000\n", argv[0]);
    printf("  >  %s -v -e entrada.dat -s resuldado.dat -m 0 -d 5000\n", argv[0]);
    exit(0);
}

void printInitParam() {
    printf("Processando com os parametros:\n");
    printf("\t Arquivo de entrada: %s\n", narq_e);
    printf("\t Arquivo de resultados: %s\n", narq_s);
    printf("\t Verbosity: %d\n", verbosity);
    printf("\t Modo (0 = Singlecore | 1 = Multicore | 2 = CUDA): %d\n", modo);
    printf("\t Threads OpenMP: %d\n", chunk);
    printf("\t Threads Cuda: %d\n", nthreads);
    printf("\t Problema aumentado em: %d\n", multiplicador);
}

void printInitMoc() {
    printf("Dados Iniciais:\n");
    printf("\tElementos da matriz = %d\n", N);
    printf("\tlinha = %d\n", linha);
    printf("\ttmax = %4.2f\n", tmax);
    printf("\tdt = %4.2f\n", dt);
    printf("\tdx = %4.2f\n", dx);
    printf("\tl = %f\n", l);
    printf("\tn = %d\n", n);
    printf("\tns = %d\n", ns);
    printf("\tN = %d\n", N);  
}


//Grava o resultado obtido no arquivo definido via paramêtro
void gravaHQTDisco(struct _HQT *matrizHQT, int tamanho) {
    if (gravarDisco == 0) {
        return;
    }
    FILE *arq_s;    // Arquivo de Saída
    if (verbosity) printf("Abrindo %s..\n", narq_s);
    arq_s = fopen(narq_s, "w"); 

    if (arq_s == NULL) {
        if (verbosity) perror ("Erro ao abrir arquivo de saida");
        #ifndef LINUX
        if (verbosity) printf( "Value of errno: %d\n", errno );
        #endif
    } else {
        if (verbosity) printf("Gravando saída\n");
        int i;
        for(i = 0; i < tamanho; i++) {
            fprintf(arq_s, "%4.2f      %4.5f      %4.5f\n", matrizHQT[i].T, matrizHQT[i].H, matrizHQT[i].Q);
        }

        fclose(arq_s);
    }
    gravarDisco = 0; //Só preciso gravar 1 resultado
}

__device__ void printConstantes(ConstantesPrograma *ct) {
    printf("Constantes no DEVICE: \n");
    printf("\tns %d", ct->ns);
    printf("\tqo %4.5f", ct->qo);
    printf("\thr %4.5f", ct->hr);
    printf("\tr %4.5f", ct->r);
    printf("\tb %4.5f", ct->b);
    printf("\tt %4.5f", ct->t);
    printf("\tdt %4.5f", ct->dt);
    printf("\ttmax %4.5f", ct->tmax);
    printf("\tcontrole %d", ct->controle);
    printf("\tnthreads %d", ct->nthreads);

}

//Cálculo do MOC (single core)
void calc_HQ_CPU();

//Cálculo do MOC (multi core)
void calc_HQ_MCPU();

//Cálculo do MOC (GPU)
__global__ void calc_HQ_RegimeTransiente(_HQT *_dMatrizHQT);
void calc_HQ_GPU();

int main(int argc, char* argv[]) {

    //Dá pau pra mostrar o tempo
    //setlocale(LC_ALL, "pt_BR.UTF-8");
    
    char str[100];       //Buffer

    float tcpu, tfunc;
    struct timeval p1start, p1stop, totalt;
    struct timeval k1start, k1stop, ktotalt;
    
    narq_e = narqe_default;
    narq_s = narqs_default;
    gravarDisco = 0;
    chunk = 512;
    verbosity = 0;
    modo = SINGLE;
    multiplicador = 1;
    nthreads = 512;
    paralelo = 0;
    exibe_tempos = 0;

    char c;
    int temp_mode = 0;
    while ( (c = getopt (argc, argv, "e:s:m:c:t:d:pvhk")) != -1) {
        
        switch(c) {
            case 'e':
                narq_e = optarg;
                break;
            case 's':
                narq_s = optarg;
                gravarDisco = 1;
                break;
            case 'm':
                temp_mode = atoi(optarg);
                if (temp_mode > 2) {
                    printUsage(argv);
                    exit(1);
                }
                break;
            case 't':
                nthreads = atoi(optarg);
                chunk = nthreads;
                break;
            case 'd':
                multiplicador = atoi(optarg);
                break;
            case 'v':
                verbosity = 1;
                break;
            case 'k':
                exibe_tempos = 1;
                break;
            case 'p':
                paralelo = 1;
                break;
            case 'h':
                printUsage(argv);
                exit(0);
            default:
                printUsage(argv);
                exit(1);
        }

    }
    
    if (temp_mode > 0 && temp_mode < 3) {
        modo = MULTI;
        if (temp_mode > 1) 
            modo = CUDA;
    }
        
    if (verbosity) printInitParam();

    if (verbosity)
        printf("Inicio do processamento\n");

    //Teste
    //Leitura dos Dados
    FILE *arq_e;    // Arquivo de Entrada        
    arq_e = fopen(narq_e, "r");
    if (arq_e == NULL) {
        if (verbosity) {
            perror ("Erro ao abrir arquivo");
            #ifndef LINUX
            printf( "Value of errno: %d\n", errno );
            #endif
        }
        exit(1);
    } else {
        //fscanf(arq_e, "%s", str);
        
        fscanf(arq_e, "%f", &hr);
            fscanf(arq_e, "%s", str);
        fscanf(arq_e, "%f", &f);
            fscanf(arq_e, "%s", str);
        fscanf(arq_e, "%f", &l);
            fscanf(arq_e, "%s", str);
        fscanf(arq_e, "%f", &d);
            fscanf(arq_e, "%s", str);            
        fscanf(arq_e, "%d", &n);
            fscanf(arq_e, "%s", str);  
        fscanf(arq_e, "%f", &tmax);
            fscanf(arq_e, "%s", str);              
        fscanf(arq_e, "%f", &a);
            fscanf(arq_e, "%s", str);

        fscanf(arq_e, "%f", &cdao);
        fclose(arq_e);    
    }   

    if (verbosity) {
        printf("Carregou o arquivo com os dados:\n");
        printf("\thr = %4.2f\n", hr);
        printf("\tf = %4.2f\n", f);
        printf("\t(comprimento) l = %4.2f\n", l);
        printf("\t(diametro) d = %4.5f\n", d);
        printf("\t(Segmentos) n = %d\n", n);
        printf("\t(Tempo Max) tmax =  %4.2f\n", tmax);
        printf("\t(celeridade) a = %4.2f\n", a);
        printf("\tcdao = %4.5f\n", cdao);
    }

    //Limpa Contadores CPU
    timerclear(&p1start); 
    timerclear(&p1stop); 

    gettimeofday(&p1start, NULL);

    int cont;
    timerclear(&k1start); 
    timerclear(&k1stop); 

    gettimeofday(&k1start, NULL);
    for (cont = 0; cont < multiplicador; cont++) {
        //printf("Rodando %d modo %d\n", cont, modo);
        switch(modo) {
            case SINGLE:
                calc_HQ_CPU();  
                break;
            case MULTI:
                calc_HQ_MCPU();    
                break;
            case CUDA:
                //printf("Cuda %d\n", cont);
                calc_HQ_GPU();    
                break;
    
        }
    
    }
    gettimeofday(&k1stop, NULL);
    timersub(&k1stop, &k1start, &ktotalt);
    tfunc = 0.001 * (ktotalt.tv_sec * 1000000 + ktotalt.tv_usec);

    if (exibe_tempos)
        printf("Tempo transitorio = %f ms\n", tfunc);

    gettimeofday(&p1stop, NULL);

    //Gasto de tempo ms
    timersub(&p1stop, &p1start, &totalt);

    tcpu = 0.001 * (totalt.tv_sec * 1000000 + totalt.tv_usec);

    if (exibe_tempos)
        printf("=== Tempo total de execução = %f ms, Modo %d ===\n", tcpu, modo);

}

void calc_HQ_CPU() {
    int i;
    int controle;       //Controle para linkar vetores H e Q as matrizes (Controla o pulo para as linhas)

    // Constantes //
    as = (PI * d * d) / 4;
    dx = l / n;
    dt = dx / a;
    ns = n + 2; //segmentos + montante + justrante
    r = f * dx / ( 2 * G * d * as * as);
    b = a/(G * as);

    //Alocação Vetor e Matrizes
    linha = (int)(tmax / dt);
    N = ns + (ns * linha); //Delta T para 0 + conjunhto de delta T + ultima iteração Delta T

    if (verbosity) printInitMoc();

    controle = 0;

    size_t sizeMATRIX = N; //* sizeof(float);
    size_t sizeVECTOR = ns; //* sizeof(float);

    matrizHQT = (struct _HQT *)malloc(sizeof(struct _HQT) * sizeMATRIX);
    
    atualHQ = (struct _HQ *)malloc(sizeof(struct _HQ) * sizeVECTOR);
    proximolHQ = (struct _HQ *)malloc(sizeof(struct _HQ) * sizeVECTOR);

    // Inicializa H e Q

    initZeroHQT(matrizHQT , sizeMATRIX);

    initZeroHQ(atualHQ, sizeVECTOR);
    initZeroHQ(proximolHQ, sizeVECTOR);

    t = 0;

    //Cálculo do Regime Permanente inicial
    if (verbosity) printf("Regime Permanente inicial\n");
    qo = sqrt(((cdao * cdao) * 2 * hr * hr)/(1 + (cdao * cdao) * 2 * G * n * r));

    for(i = 0; i < ns; i++) {
        atualHQ[i].H = hr - (i) * r * qo * qo;
        atualHQ[i].Q = qo;
        matrizHQT[i].H = atualHQ[i].H;
        matrizHQT[i].Q = atualHQ[i].Q;
        matrizHQT[i].T = t;    
    }

    //Inicio Transitório
    while (t < tmax) {
        t = t + dt;
        controle++;

        //Pontos interiores;
        for(i = 1; i < ns; i++) {
            cp = atualHQ[i-1].H + b * atualHQ[i-1].Q;
            bp = b + r * fabs(atualHQ[i-1].Q);
            cm = atualHQ[i+1].H - b * atualHQ[i+1].Q;
            bm = b + r * fabs(atualHQ[i+1].Q);
            proximolHQ[i].Q = (cp - cm) / (bp + bm);
            proximolHQ[i].H = cp - bp * proximolHQ[i].Q;
        }

        //Condição de contorno de montante: Reservatório

        proximolHQ[0].H = hr;
        cm = atualHQ[1].H - b * atualHQ[1].Q;
        bm = b + r * fabs(atualHQ[1].Q);
        proximolHQ[0].Q = (hr - cm)/bm;

        //condição de contorno de jusante: Válvula

        cp = atualHQ[ns-2].H + b * atualHQ[ns-2].Q;
        proximolHQ[ns-1].Q = 0;
        proximolHQ[ns-1].H = cp;

        //Atualização

        for (i = 0; i < ns; i++) {
            atualHQ[i].H = proximolHQ[i].H;
            atualHQ[i].Q = proximolHQ[i].Q;   
            
            unsigned int ind = (controle * ns) + i;
            if (ind < sizeMATRIX) {
                matrizHQT[ind].H = atualHQ[i].H;
                matrizHQT[ind].Q = atualHQ[i].Q;
                matrizHQT[ind].T = t; 
            }
        }

    }

    if (verbosity) printf("Processamento Completo\n");
    //Impressão completa
    gravaHQTDisco(matrizHQT, sizeMATRIX);
    
    free(matrizHQT);

    free(atualHQ);
    free(proximolHQ);
}

void calc_HQ_MCPU() {
    int i;
    int linha;
    int controle;       //Controle para linkar vetores H e Q as matrizes (Controla o pulo para as linhas)

    // Constantes //
    as = (PI * d * d) / 4;
    dx = l / n;
    dt = dx / a;
    ns = n + 2; //segmentos + montante + justrante
    r = f * dx / ( 2 * G * d * as * as);
    b = a/(G * as);

    //Alocação Vetor e Matrizes
    //linha = ((int)(tmax/dt))+1;;
    //N = ns + (ns * linha)+ns; //Delta T para 0 + conjunhto de delta T + ultima iteração Delta T
    linha = (int)(tmax / dt);  
    N = ns + (ns * linha);

    if (verbosity) printInitMoc();

    controle = 0;

    size_t sizeMATRIX = N;
    size_t sizeVECTOR = ns;

    matrizHQT = (struct _HQT *)malloc(sizeof(struct _HQT) * sizeMATRIX);
    
    atualHQ = (struct _HQ *)malloc(sizeof(struct _HQ) * sizeVECTOR);
    proximolHQ = (struct _HQ *)malloc(sizeof(struct _HQ) * sizeVECTOR);

    // Inicializa H e Q

    initZeroHQT(matrizHQT , sizeMATRIX);

    initZeroHQ(atualHQ, sizeVECTOR);
    initZeroHQ(proximolHQ, sizeVECTOR);

    t = 0;

    //Cálculo do Regime Permanente inicial
    if (verbosity) printf("Regime Permanente inicial\n");
    qo = sqrt(((cdao * cdao) * 2 * hr * hr)/(1 + (cdao * cdao) * 2 * G * n * r));

    #pragma omp parallel shared(atualHQ,matrizHQT,hr,r,qo,t,ns,chunk) private(i) 
    {
        #pragma omp for schedule(dynamic,chunk) nowait
            for(i = 0; i < ns; i++) {
                atualHQ[i].H = hr - (i) * r *qo * qo;
                atualHQ[i].Q = qo;
                matrizHQT[i].H = atualHQ[i].H;
                matrizHQT[i].Q = atualHQ[i].Q;
                matrizHQT[i].T = t;    
            }
    }

    //Inicio Transitório
    while (t < tmax) {
        if (verbosity) printf("Passo T %f de %f\n", t, tmax);
        t = t + dt;
        controle++;

        #pragma omp parallel shared(atualHQ,proximolHQ,r,b,ns,chunk) private(i,cm,bm,cp,bp) 
        {
            #pragma omp for schedule(dynamic,chunk) nowait
                //Pontos interiores;
                for(i = 1; i < ns; i++) {
                    cp = atualHQ[i-1].H + b * atualHQ[i-1].Q;
                    bp = b + r * fabs(atualHQ[i-1].Q);
                    cm = atualHQ[i+1].H - b * atualHQ[i+1].Q;
                    bm = b + r * fabs(atualHQ[i+1].Q);
                    proximolHQ[i].Q = (cp - cm) / (bp + bm);
                    proximolHQ[i].H = cp - bp * proximolHQ[i].Q;
                }
        }
        //Condição de contorno de montante: Reservatório

        proximolHQ[0].H = hr;
        cm = atualHQ[1].H - b * atualHQ[1].Q;
        bm = b + r * fabs(atualHQ[1].Q);
        proximolHQ[0].Q = (hr - cm)/bm;

        //condição de contorno de jusante: Válvula

        cp = atualHQ[ns-2].H + b * atualHQ[ns-2].Q;
        proximolHQ[ns-1].Q = 0;
        proximolHQ[ns-1].H = cp;

        //Atualização
        #pragma omp parallel shared(atualHQ,proximolHQ,matrizHQT,ns,t,chunk) private(i) 
        {
            #pragma omp for schedule(dynamic,chunk) nowait
                for (i = 0; i < ns; i++) {
                    atualHQ[i].H = proximolHQ[i].H;
                    atualHQ[i].Q = proximolHQ[i].Q;                    
                    unsigned int ind = (controle * ns) + i;
                    if (ind < sizeMATRIX) {
                        matrizHQT[ind].H = atualHQ[i].H;
                        matrizHQT[ind].Q = atualHQ[i].Q;
                        matrizHQT[ind].T = t; 
                    }
            
                }
        }        
    }

    if (verbosity) printf("Processamento Completo\n");
    //Impressão completa
    gravaHQTDisco(matrizHQT, sizeMATRIX);
    
    free(matrizHQT);
    
    free(atualHQ);
    free(proximolHQ);
}

void calc_HQ_GPU() {
    //int linha;
    int controle;       //Controle para linkar vetores H e Q as matrizes (Controla o pulo para as linhas)

    // Constantes //
    as = (PI * d * d) / 4;
    dx = l / n;
    dt = dx / a;
    ns = n + 2; //segmentos + montante + justrante
    r = f * dx / ( 2 * G * d * as * as);
    b = a/(G * as);

    //Alocação Vetor e Matrizes
    linha = (int)(tmax / dt);
    N = ns + (ns * linha); //Delta T para 0 + conjunhto de delta T + ultima iteração Delta T

    //printf("Uso da memória %d para N=%d \n", (int)(sizeof(struct _HQT) * N), N);
    if (verbosity) printInitMoc();

    controle = 0;

    size_t sizeMATRIX = N; //* sizeof(float);

    matrizHQT = (_HQT *)malloc(sizeof(_HQT) * sizeMATRIX);
    
    // Inicializa H e Q
    initZeroHQT(matrizHQT , sizeMATRIX);
    //Aloca vetores na memória do device
    HANDLE_ERROR( hipMallocManaged(&_dMatrizHQT, (sizeof(_HQT) * sizeMATRIX)) );

    //Dados : Host -> Device
    HANDLE_ERROR( hipMemcpy(_dMatrizHQT, matrizHQT, (sizeof(_HQT) * sizeMATRIX), hipMemcpyHostToDevice) );
    
    //Cálculo do Regime Permanente inicial
    t = 0;
    qo = sqrt(((cdao * cdao) * 2 * hr * hr)/(1 + (cdao * cdao) * 2 * G * n * r));
 
    nblocks = (ns + nthreads) / nthreads;
    
    //no c++ daria pra passar um objeto ?
    ConstantesPrograma *host_constantes = (ConstantesPrograma *)malloc(sizeof(ConstantesPrograma));
    host_constantes->ns = ns;
    host_constantes->hr = hr;
    host_constantes->r = r;
    host_constantes->b = b;
    host_constantes->dt = dt;
    host_constantes->tmax = tmax;
    host_constantes->controle = controle;
    host_constantes->t = t;
    host_constantes->qo = qo;
    host_constantes->nthreads = nthreads;

    //HANDLE_ERROR( hipMallocManaged(&ConstantDeviceConstantes, sizeof(ConstantesPrograma)) );
    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(ConstantDeviceConstantes), host_constantes, sizeof(ConstantesPrograma)) );
    free(host_constantes);

    hipEvent_t start, stop;

    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    calc_HQ_RegimeTransiente<<<nblocks, nthreads, nthreads*sizeof(_HQ)>>>(_dMatrizHQT);

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	
	float elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
    
    if (exibe_tempos)
        printf("Tempo do Kernel = %f ms\n", elapsedTime);
    
    //Dados : Device -> Host
    HANDLE_ERROR( hipMemcpy(matrizHQT, _dMatrizHQT, (sizeof(_HQT) * sizeMATRIX), hipMemcpyDeviceToHost) );

    if (verbosity) printf("Processamento Completo\n");
    //Impressão completa
    gravaHQTDisco(matrizHQT, sizeMATRIX);

    free(matrizHQT);
    hipFree(_dMatrizHQT);
}

__global__ void calc_HQ_RegimeTransiente(_HQT *_dMatrizHQT) {
    /*
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    */

    float _cp = 0.0;
    float _bp = 0.0;
    float _cm = 0.0;
    float _bm = 0.0;  

    unsigned int _ns = ConstantDeviceConstantes.get_ns();
    float _qo = ConstantDeviceConstantes.get_qo();
    float _hr = ConstantDeviceConstantes.get_hr();
    float _r = ConstantDeviceConstantes.get_r();
    float _b = ConstantDeviceConstantes.get_b();
    float _t = ConstantDeviceConstantes.get_t();
    float _dt = ConstantDeviceConstantes.get_dt();
    float _tmax = ConstantDeviceConstantes.get_tmax();
    unsigned int _nthreads = ConstantDeviceConstantes.get_nthreads();

    unsigned int _controle = 0;  

    //printConstantes(&ConstantDeviceConstantes);

    int _i = threadIdx.x + blockIdx.x * blockDim.x;
    int indice = (int)(_i % _nthreads);
  
    extern __shared__ _HQ tempoAtual[]; 
    extern __shared__ _HQ tempoProximo[]; 

    //Regime Permanente inicial
    if (_i < _ns) {
        tempoAtual[indice].H = _hr - (_i) * _r *_qo * _qo;
        tempoAtual[indice].Q = _qo;

        __syncthreads();

        _dMatrizHQT[(_controle * _ns) + _i].H = tempoAtual[indice].H;
        _dMatrizHQT[(_controle * _ns) + _i].Q = tempoAtual[indice].Q;
        _dMatrizHQT[(_controle * _ns) + _i].T = _t;  
    }

    //Regime Transitório
    if (_i < _ns) {

        while (_t < _tmax) {
            _t = _t + _dt;
            _controle++;

            //Pontos interiores;
            if (_i < _ns && _i >= 1) {
                _cp = tempoAtual[indice-1].H + _b * tempoAtual[indice-1].Q;
                _bp = _b + _r * fabs(tempoAtual[indice-1].Q);
                _cm = tempoAtual[indice+1].H - _b * tempoAtual[indice+1].Q;
                _bm = _b + _r * fabs(tempoAtual[indice+1].Q);
                tempoProximo[indice].Q = (_cp - _cm) / (_bp + _bm);
                tempoProximo[indice].H = _cp - _bp * tempoProximo[indice].Q;
            }

            //Condição de contorno de montante: Reservatório
            if (_i == 0) {
                tempoProximo[0].H = _hr;
                _cm = tempoAtual[1].H - _b * tempoAtual[1].Q;
                _bm = _b + _r * fabs(tempoAtual[1].Q);
                tempoProximo[0].Q = (_hr - _cm)/_bm;
            }


            //condição de contorno de jusante: Válvula
            if (_i == _ns - 1) {
                _cp = tempoAtual[_ns-2].H + _b * tempoAtual[_ns-2].Q;
                tempoProximo[_ns-1].Q = 0;
                tempoProximo[_ns-1].H = _cp;
            }

            //Atualização

            tempoAtual[indice].H = tempoProximo[indice].H;
            tempoAtual[indice].Q = tempoProximo[indice].Q;

            __syncthreads();

            _dMatrizHQT[(_controle * _ns) + _i].H = tempoAtual[_i].H;
            _dMatrizHQT[(_controle * _ns) + _i].Q = tempoAtual[_i].Q;
            _dMatrizHQT[(_controle * _ns) + _i].T = _t;

        }
    }
}
